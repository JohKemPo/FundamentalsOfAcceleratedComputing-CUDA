#include "hip/hip_runtime.h"

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include "files.h"

#define SOFTENING 1e-9f

typedef struct
{ 
    float x, y, z, vx, vy, vz; 
} Body;

__global__ void bodyForce(Body *p, float dt, int n) //Tornando a fun GPU
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    //Exclui o for inicial 
    
    if (i < n) 
    {
        float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;

        for (int j = 0; j < n; j++) 
        {
            float dx = p[j].x - p[i].x;
            float dy = p[j].y - p[i].y;
            float dz = p[j].z - p[i].z;
            float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
            float invDist = rsqrtf(distSqr);
            float invDist3 = invDist * invDist * invDist;
            
            Fx += dx * invDist3; Fy += dy * invDist3; Fz += dz * invDist3;
        }
        p[i].vx += dt*Fx; p[i].vy += dt*Fy; p[i].vz += dt*Fz;
    }
}

int main(const int argc, const char** argv) 
{
    int nBodies = 2<<11;
    

    if (argc > 1)nBodies = 2<<atoi(argv[1]);

    const char * initialized_values;
    const char * solution_values;

    if (nBodies == 2<<11) 
    {
        initialized_values = "09-nbody/files/initialized_4096";
        solution_values = "09-nbody/files/solution_4096";
    } 
    else // nBodies == 2<<15
    {
        initialized_values = "09-nbody/files/initialized_65536";
        solution_values = "09-nbody/files/solution_65536";
    }

    if (argc > 2) 
        initialized_values = argv[2];
    if (argc > 3) 
        solution_values = argv[3];

    const float dt = 0.01f;
    const int nIters = 10;

    hipError_t bodyForceErr;
    hipError_t asyncErr;

    int DeviceNum;
    int QuantidadeSmi;
    
    hipGetDevice(&DeviceNum);
    hipDeviceGetAttribute(&QuantidadeSmi, hipDeviceAttributeMultiprocessorCount, DeviceNum);

    int NumeroDeThreads = 128;
    int NumeroDeBlocos = 32 * QuantidadeSmi;

    float *buf;
    int bytes = nBodies*sizeof(Body);

    hipMallocManaged(&buf, bytes);//MallocCUDA
    Body *p = (Body*)buf;

    read_values_from_file(initialized_values, buf, bytes);

    double totalTime = 0.0; 
    for (int iter = 1; iter <= nIters; iter++) 
    {
        StartTimer();

        bodyForce<<<NumeroDeBlocos, NumeroDeThreads>>>(p, dt, nBodies);

        bodyForceErr = hipGetLastError();
        if(bodyForceErr != hipSuccess) 
            printf("Error: %s\n", hipGetErrorString(bodyForceErr));
    
        asyncErr = hipDeviceSynchronize();
        if(asyncErr != hipSuccess) 
            printf("Error: %s\n", hipGetErrorString(asyncErr));

        for (int i = 0 ; i < nBodies; i++) 
        { // integrate position
            p[i].x += p[i].vx*dt;
            p[i].y += p[i].vy*dt;
            p[i].z += p[i].vz*dt;
        }

        const double tElapsed = GetTimer() / 1000.0;
        if (iter > 1)
            totalTime += tElapsed; 
    }
    double avgTime = totalTime / (double)(nIters-1); 

    float billionsOfOpsPerSecond = 1e-9 * nBodies * nBodies / avgTime;
    write_values_to_file(solution_values, buf, bytes);

    printf("%0.3f Billion Interactions / second\n", billionsOfOpsPerSecond);
    
    hipFree(buf);//FREECUDA
}